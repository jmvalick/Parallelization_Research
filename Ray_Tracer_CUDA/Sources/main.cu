#include "hip/hip_runtime.h"
#define MAXFLOAT 9999999

#include <stdlib.h>
#include <iostream>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "fileCreate.h"
#include "hitable_list.cuh"
#include "sphere.cuh"
#include "camera.cuh"
#include "material.cuh"
#include "vec3.cuh"

using namespace std;



///     *** CUDA Version ***    ///
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__device__ vec3 color_cu(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0f, 0.0f, 0.0f);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *image, int max_x, int max_y, int samples, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color_cu(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(samples);
    col = vec3( sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
    image[pixel_index] = 255.99f * col;

    /*if ((max_y - j) % 25 == 0) {
        printf("Current Vertical Pixel: %d\n", max_y - j);
    }*/
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int image_width, int image_height, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + 0.9f * RND, 0.2, b + 0.9f * RND);
                if (choose_mat < 0.3f) {  // diffuse
                    d_list[i++] = new sphere(center, 0.2f, new lambertian(vec3(RND, RND, RND)));
                }
                else if (choose_mat < 0.60f) {
                    d_list[i++] = new sphere(center, 0.2f, new metal(vec3(RND, RND, RND), 0.5f * RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2f, new dielectric(vec3(RND, RND, RND), 1.5f));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(vec3(RND, RND, RND), 1.5f));
        d_list[i++] = new sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(vec3(RND, RND, RND)));
        d_list[i++] = new sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(vec3(RND, RND, RND), 0.5f * RND));
        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_list, 22 * 22 + 1 + 3);

        // Initialize Camera
        vec3 lookfrom(13.0f, 2.0f, 3.0f);
        vec3 lookat(0.0f, 0.0f, 0.0f);
        float dist_to_focus = 10.0f;
        float aperture = 0.1f;
        *d_camera = new camera(lookfrom, lookat, vec3(0.0f, 1.0f, 0.0f), 30.0f, float(image_width)/float(image_height), aperture, dist_to_focus);
    }
}


__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}


int main() {

    
    // Screen size and samples data
    int image_dim_list[8][2] = { {256, 144}, {426, 240}, {640, 360}, {854, 480}, {1280, 720}, {1920, 1080}, {2560, 1440}, {3840, 2160} };
    int samples_list[8] = { 1, 2, 4, 8, 16, 32, 64, 128 };

    //int image_dim_list[3][2] = { {256, 144}, {426, 240}, {640, 360} };
    //int samples_list[4] = { 1, 2, 4, 8 };

    const int num_resolutions = sizeof(image_dim_list) / sizeof(image_dim_list[0]);
    const int num_samples = sizeof(samples_list) / sizeof(samples_list[0]);


    double** time_list = (double**)malloc(num_resolutions * sizeof(double*));
    for (int i = 0; i < num_resolutions; ++i) {
        time_list[i] = (double*)malloc(num_samples * sizeof(double));
    }

    const int block_width = 16;
    const int block_height = 16;


    int count = 0;
    for (int i = 0; i < (num_resolutions); i++) {
        for (int j = 0; j < (num_samples); j++) {

            const int image_width = image_dim_list[i][0];
            const int image_height = image_dim_list[i][1];
            const int samples = samples_list[j];

            std::cout << "Image size: " << image_width << "x" << image_height << endl;
            std::cout << "Samples per pixel: " << samples << endl;
            std::cout << "Device: GPU" << endl;
            std::cout << "Block width: " << block_width << endl;
            std::cout << "Block height: " << block_height << endl;

            // Allocate frame buffer
            int num_pixels = image_width * image_height;
            size_t image_size = num_pixels * sizeof(vec3);
            vec3* image;
            checkCudaErrors(hipMallocManaged((void**)&image, image_size));

            // Allocate Random State
            hiprandState* d_rand_state;
            checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
            hiprandState* d_rand_state2;
            checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

            rand_init <<<1, 1 >>> (d_rand_state2);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());

            // Allocate scene
            hitable** d_list;
            int num_hitables = 22 * 22 + 1 + 3;
            checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hitable*)));
            hitable** d_world;
            checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
            camera** d_camera;
            checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
            create_world <<<1, 1 >>> (d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());

            // Timer
            clock_t start, stop;
            start = clock();

            // Render image
            dim3 blocks(image_width / block_width + 1, image_height / block_height + 1);
            dim3 threads(block_width, block_height);
            render_init <<<blocks, threads >>> (image_width, image_height, d_rand_state);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            render <<<blocks, threads >>> (image, image_width, image_height, samples, d_camera, d_world, d_rand_state);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());

            stop = clock();
            double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
            time_list[i][j] = timer_seconds;
            std::cout << "\nRendered time: " << timer_seconds << " seconds\n\n";


            // Export to ppm
            createPPMFile(image, image_width, image_height, samples);

            // Clean up
            checkCudaErrors(hipDeviceSynchronize());
            free_world <<<1, 1 >>> (d_list, d_world, d_camera);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipFree(d_camera));
            checkCudaErrors(hipFree(d_world));
            checkCudaErrors(hipFree(d_list));
            checkCudaErrors(hipFree(d_rand_state));
            checkCudaErrors(hipFree(d_rand_state2));
            checkCudaErrors(hipFree(image));

            hipDeviceReset();
        }
    }

    createTextFile(image_dim_list, num_resolutions, samples_list, num_samples, time_list);

    return 0;
}