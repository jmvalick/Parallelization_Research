#include "hitable_list.cuh"


__device__ bool hitable_list::hit(const ray& r, float tmin, float tmax, hit_record& rec) const {
	hit_record temp_rec;
	bool hit_anything = false;
	float closest = tmax;
	for (int i = 0; i < list_size; i++) {
		if (list[i]->hit(r, tmin, closest, temp_rec)) {
			hit_anything = true;
			closest = temp_rec.t;
			rec = temp_rec;
		}
	}
	return hit_anything;
}